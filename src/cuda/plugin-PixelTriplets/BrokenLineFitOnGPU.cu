#include "BrokenLineFitOnGPU.h"
#include "CUDACore/device_unique_ptr.h"

void HelixFitOnGPU::launchBrokenLineKernels(HitsView const *hv,
                                            uint32_t hitsInFit,
                                            uint32_t maxNumberOfTuples,
                                            hipStream_t stream) {
  assert(tuples_d);

  //  Fit internals
  auto hitsGPU_ = cms::cuda::make_device_unique<double[]>(
      maxNumberOfConcurrentFits_ * sizeof(Rfit::Matrix3xNd<4>) / sizeof(double), stream);
  auto hits_geGPU_ = cms::cuda::make_device_unique<float[]>(
      maxNumberOfConcurrentFits_ * sizeof(Rfit::Matrix6x4f) / sizeof(float), stream);
  auto fast_fit_resultsGPU_ = cms::cuda::make_device_unique<double[]>(
      maxNumberOfConcurrentFits_ * sizeof(Rfit::Vector4d) / sizeof(double), stream);

  cms::cuda::ExecutionConfiguration exec;
  auto blockSize_ff3 = exec.configFromFile("kernelFastFit3");
  auto numberOfBlocks_ff3 = (maxNumberOfConcurrentFits_ + blockSize_ff3 - 1) / blockSize_ff3;

  auto blockSize_blf3 = exec.configFromFile("kernelLineFit3");
  auto numberOfBlocks_blf3 = (maxNumberOfConcurrentFits_ + blockSize_blf3 - 1) / blockSize_blf3;

  auto blockSize_ff4 = exec.configFromFile("kernelFastFit4");
  auto numberOfBlocks_ff4 = (maxNumberOfConcurrentFits_ + blockSize_ff4 - 1) / blockSize_ff4;

  auto blockSize_blf4 = exec.configFromFile("kernelLineFit4");
  auto numberOfBlocks_blf4 = (maxNumberOfConcurrentFits_ + blockSize_blf4 - 1) / blockSize_blf4;

  auto blockSize_ff5 = exec.configFromFile("kernelFastFit5");
  auto numberOfBlocks_ff5 = (maxNumberOfConcurrentFits_ + blockSize_ff5 - 1) / blockSize_ff5;

  auto blockSize_blf5 = exec.configFromFile("kernelLineFit5");
  auto numberOfBlocks_blf5 = (maxNumberOfConcurrentFits_ + blockSize_blf5 - 1) / blockSize_blf5;
  for (uint32_t offset = 0; offset < maxNumberOfTuples; offset += maxNumberOfConcurrentFits_) {
    // fit triplets
    kernelBLFastFit<3><<<numberOfBlocks_ff3, blockSize_ff3, 0, stream>>>(
        tuples_d, tupleMultiplicity_d, hv, hitsGPU_.get(), hits_geGPU_.get(), fast_fit_resultsGPU_.get(), 3, offset);
    cudaCheck(hipGetLastError());

    kernelBLFit<3><<<numberOfBlocks_blf3, blockSize_blf3, 0, stream>>>(tupleMultiplicity_d,
                                                             bField_,
                                                             outputSoa_d,
                                                             hitsGPU_.get(),
                                                             hits_geGPU_.get(),
                                                             fast_fit_resultsGPU_.get(),
                                                             3,
                                                             offset);
    cudaCheck(hipGetLastError());

    // fit quads
    kernelBLFastFit<4><<<numberOfBlocks_ff4 / 4, blockSize_ff4, 0, stream>>>(
        tuples_d, tupleMultiplicity_d, hv, hitsGPU_.get(), hits_geGPU_.get(), fast_fit_resultsGPU_.get(), 4, offset);
    cudaCheck(hipGetLastError());

    kernelBLFit<4><<<numberOfBlocks_blf4 / 4, blockSize_blf4, 0, stream>>>(tupleMultiplicity_d,
                                                                 bField_,
                                                                 outputSoa_d,
                                                                 hitsGPU_.get(),
                                                                 hits_geGPU_.get(),
                                                                 fast_fit_resultsGPU_.get(),
                                                                 4,
                                                                 offset);
    cudaCheck(hipGetLastError());

    if (fit5as4_) {
      // fit penta (only first 4)
      kernelBLFastFit<4><<<numberOfBlocks_ff4 / 4, blockSize_ff4, 0, stream>>>(
          tuples_d, tupleMultiplicity_d, hv, hitsGPU_.get(), hits_geGPU_.get(), fast_fit_resultsGPU_.get(), 5, offset);
      cudaCheck(hipGetLastError());

      kernelBLFit<4><<<numberOfBlocks_blf4 / 4, blockSize_blf4, 0, stream>>>(tupleMultiplicity_d,
                                                                   bField_,
                                                                   outputSoa_d,
                                                                   hitsGPU_.get(),
                                                                   hits_geGPU_.get(),
                                                                   fast_fit_resultsGPU_.get(),
                                                                   5,
                                                                   offset);
      cudaCheck(hipGetLastError());
    } else {
      // fit penta (all 5)
      kernelBLFastFit<5><<<numberOfBlocks_ff5 / 4, blockSize_ff5, 0, stream>>>(
          tuples_d, tupleMultiplicity_d, hv, hitsGPU_.get(), hits_geGPU_.get(), fast_fit_resultsGPU_.get(), 5, offset);
      cudaCheck(hipGetLastError());

      kernelBLFit<5><<<numberOfBlocks_blf5 / 4, blockSize_blf5, 0, stream>>>(tupleMultiplicity_d,
                                                                   bField_,
                                                                   outputSoa_d,
                                                                   hitsGPU_.get(),
                                                                   hits_geGPU_.get(),
                                                                   fast_fit_resultsGPU_.get(),
                                                                   5,
                                                                   offset);
      cudaCheck(hipGetLastError());
    }

  }  // loop on concurrent fits
}
